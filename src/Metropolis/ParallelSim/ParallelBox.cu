/*
	Implements methods related to managing data between the host and device.
	Subclass of Box.

	Created: February 21, 2014
	
	-> February 26, by Albert Wallace
	-> March 28, by Joshua Mosby
	-> April 21, by Nathan Coleman
*/

#include "ParallelBox.cuh"

using namespace std;

ParallelBox::ParallelBox(): Box()
{
	//Is anything really needed here?
}

ParallelBox::~ParallelBox()
{
	// TODO: free device memory
}

int ParallelBox::changeMolecule(int molIdx)
{
	Box::changeMolecule(molIdx);
	writeChangeToDevice(molIdx);
	
	return molIdx;
}

int ParallelBox::rollback(int molIdx)
{
	Box::rollback(molIdx);
	writeChangeToDevice(molIdx);
	
	return molIdx;
}

void ParallelBox::copyDataToDevice()
{
	//create AtomData on host, and fill atomic data arrays on device
	atomsH = new AtomData(atoms, atomCount);
	hipMalloc(&xD, atomCount * sizeof(Real));
	hipMalloc(&yD, atomCount * sizeof(Real));
	hipMalloc(&zD, atomCount * sizeof(Real));
	hipMalloc(&sigmaD, atomCount * sizeof(Real));
	hipMalloc(&epsilonD, atomCount * sizeof(Real));
	hipMalloc(&chargeD, atomCount * sizeof(Real));
	hipMemcpy(xD, atomsH->x, atomCount * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(yD, atomsH->y, atomCount * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(zD, atomsH->z, atomCount * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(sigmaD, atomsH->sigma, atomCount * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(epsilonD, atomsH->epsilon, atomCount * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(chargeD, atomsH->charge, atomCount * sizeof(Real), hipMemcpyHostToDevice);
	
	//create device AtomData struct with pointers to filled-in atomic data arrays
	AtomData *tempAD = (AtomData*) malloc(sizeof(AtomData));
	tempAD->x = xD;
	tempAD->y = yD;
	tempAD->z = zD;
	tempAD->sigma = sigmaD;
	tempAD->epsilon = epsilonD;
	tempAD->charge = chargeD;
	tempAD->atomCount = atomsH->atomCount;
	hipMalloc(&atomsD, sizeof(AtomData));
	hipMemcpy(atomsD, tempAD, sizeof(AtomData), hipMemcpyHostToDevice);
	
	//create MoleculeData on host, and fill molecular data arrays on device
//	printf("TotalPrimaryIndexSize: %d\n", moleculesH->totalPrimaryIndexSize);
	moleculesH = new MoleculeData(molecules, moleculeCount, environment);
	hipMalloc(&atomsIdxD, moleculeCount * sizeof(int));
	hipMalloc(&numOfAtomsD, moleculeCount * sizeof(int));
	hipMalloc(&typeD, moleculeCount * sizeof(int));
	hipMalloc(&primaryIndexesD, moleculesH->totalPrimaryIndexSize * sizeof(int));
	hipMemcpy(atomsIdxD, moleculesH->atomsIdx, moleculeCount * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(numOfAtomsD, moleculesH->numOfAtoms, moleculeCount * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(typeD, moleculesH->type, moleculeCount * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(primaryIndexesD, moleculesH->primaryIndexes, moleculesH->totalPrimaryIndexSize * sizeof(int), hipMemcpyHostToDevice);
	
	//create device MoleculeData struct with pointers to filled-in molecular data arrays
	MoleculeData *tempMD = (MoleculeData*) malloc(sizeof(MoleculeData));
	tempMD->atomsIdx = atomsIdxD;
	tempMD->numOfAtoms = numOfAtomsD;
	tempMD->type = typeD;
	tempMD->primaryIndexes = primaryIndexesD;
	tempMD->moleculeCount = moleculesH->moleculeCount;
	tempMD->totalPrimaryIndexSize = moleculesH->totalPrimaryIndexSize;
	hipMalloc(&moleculesD, sizeof(MoleculeData));
	hipMemcpy(moleculesD, tempMD, sizeof(MoleculeData), hipMemcpyHostToDevice);
	
	//data structures for neighbor batch in energy calculation
	nbrMolsH = (int*) malloc(moleculeCount * sizeof(int));
	molBatchH = (int*) malloc(moleculeCount * sizeof(int));
	hipMalloc(&(nbrMolsD), moleculeCount * sizeof(int));
	hipMalloc(&(molBatchD), moleculeCount * sizeof(int));
	
	//upper bound on number of atoms in any molecule
	maxMolSize = 0;
	for (int i = 0; i < moleculesH->moleculeCount; i++)
	{
		if (moleculesH->numOfAtoms[i] > maxMolSize)
		{
			maxMolSize = moleculesH->numOfAtoms[i];
		}
	}
	
	//energies array on device has one segment for each molecule
	//where each segment has the maximum number of
	//possible interatomic energies for one pair of molecules
	energyCount = moleculesH->moleculeCount * maxMolSize * maxMolSize;
	hipMalloc(&(energiesD), energyCount * sizeof(Real));
	
	//initialize energies to 0
	hipMemset(energiesD, 0, energyCount * sizeof(Real));
	
	//copy Environment to device
	hipMalloc(&(environmentD), sizeof(Environment));
	hipMemcpy(environmentD, environment, sizeof(Environment), hipMemcpyHostToDevice);
}

void ParallelBox::writeChangeToDevice(int changeIdx)
{
	//update AtomData atomsH (MoleculeData will not change)
	int startIdx = moleculesH->atomsIdx[changeIdx];
	for (int i = 0; i < molecules[changeIdx].numOfAtoms; i++)
	{
		atomsH->x[startIdx + i] = molecules[changeIdx].atoms[i].x;
		atomsH->y[startIdx + i] = molecules[changeIdx].atoms[i].y;
		atomsH->z[startIdx + i] = molecules[changeIdx].atoms[i].z;
		//sigma, epsilon, and charge will not change, so there is no need to update those arrays
	}

	//copy changed atom data to device
	hipMemcpy(xD + startIdx, atomsH->x + startIdx, moleculesH->numOfAtoms[changeIdx] * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(yD + startIdx, atomsH->y + startIdx, moleculesH->numOfAtoms[changeIdx] * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(zD + startIdx, atomsH->z + startIdx, moleculesH->numOfAtoms[changeIdx] * sizeof(Real), hipMemcpyHostToDevice);
	//sigma, epsilon, and charge will not change, so there is no need to update those arrays
}
